#include "hip/hip_runtime.h"
/**
 * CUDA kernels for convolution.
 *
 * Yujia Li, 03/2015
 */

#include "cudamat_conv_kernels.cuh"

__global__ void kConvolveV1(float* image, float* filter, float* target,
        int n, int c, int im_h, int im_w, int n_ftr, int ftr_h, int ftr_w) {

    const int target_h = im_h - ftr_h + 1;
    const int target_w = im_w - ftr_w + 1;
    const int target_c_size = target_h * target_w;
    const int target_im_size = target_c_size * n_ftr;

    const int image_c_size = im_h * im_w;
    const int image_im_size = image_c_size * c;

    const int ftr_c_size = ftr_h * ftr_w;
    const int ftr_im_size = ftr_c_size * c;

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid / target_im_size < n) {
        int t_w = tid % target_w;
        int t_h = (tid % target_c_size) / target_w;
        int t_c = (tid % target_im_size) / target_c_size;
        int t_n = tid / target_im_size;

        float s = 0;

        for (int k = 0; k < c; k++)
            for (int i = 0; i < ftr_h; i++)
                for (int j = 0; j < ftr_w; j++) {
                    s += image[t_n * image_im_size + k * image_c_size + (t_h + i) * im_w + (t_w + j)] * \
                         filter[t_c * ftr_im_size + k * ftr_c_size + i * ftr_w + j];
                }

        target[tid] = s;

        tid += gridDim.x * blockDim.x;
    }
}

__global__ void kConvolveV2(float* image, float* filter, float* target,
        int n, int c, int im_h, int im_w, int n_ftr, int ftr_h, int ftr_w) {

    __shared__ float partial[CONV_SHARED_MEMORY_SIZE];
    const int target_h = im_h - ftr_h + 1;
    const int target_w = im_w - ftr_w + 1;
    const int target_c_size = target_h * target_w;
    const int target_im_size = target_c_size * n_ftr;

    const int image_c_size = im_h * im_w;
    const int image_im_size = image_c_size * c;

    const int ftr_c_size = ftr_h * ftr_w;
    const int ftr_im_size = ftr_c_size * c;
    const int ftr_full_size = ftr_im_size * n_ftr;

    // const int shared_batch_size = CONV_SHARED_MEMORY_SIZE / ftr_im_size * ftr_im_size;
    const int shared_batch_size = CONV_SHARED_MEMORY_SIZE;

    int ftr_idx = 0;
    int tid;
    int f_w, f_h, f_c, f_n;
    int t_w, t_h, t_n, t_base_idx, i_base_idx;

    while (ftr_idx < ftr_full_size) {
        // load one batch of filter
        __syncthreads();
        for (int i = threadIdx.x; i < shared_batch_size && i + ftr_idx < ftr_full_size; i += blockDim.x)
            partial[i] = filter[i + ftr_idx];
        __syncthreads();

        tid = threadIdx.x + blockIdx.x * blockDim.x;

        while (tid < target_c_size * n) {
            t_w = tid % target_w;
            t_h = (tid % target_c_size) / target_w;
            t_n = tid / target_c_size;

            t_base_idx = t_n * target_im_size + t_h * target_w + t_w;
            i_base_idx = t_n * image_im_size + t_h * im_w + t_w;

            int last_f_n = -1;
            float s = 0;

            for (int i = 0; i < shared_batch_size && i + ftr_idx < ftr_full_size; i++) {
                f_w = (i + ftr_idx) % ftr_w;
                f_h = ((i + ftr_idx) % ftr_c_size) / ftr_w;
                f_c = ((i + ftr_idx) % ftr_im_size) / ftr_c_size;
                f_n = (i + ftr_idx) / ftr_im_size;
                if (i > 0 && f_n != last_f_n) {
                    target[t_base_idx + last_f_n * target_c_size] += s;
                    s = 0;
                    last_f_n = f_n;
                }

                s += partial[i] * image[i_base_idx + f_c * image_c_size + f_h * im_w + f_w];
            }
            if (s != 0)
                target[t_base_idx + last_f_n * target_c_size] += s;

            tid += gridDim.x * blockDim.x;
        }
        ftr_idx += shared_batch_size;
    }
}


__global__ void kConvolveV3(float* image, float* filter, float* target,
        int n, int c, int im_h, int im_w, int n_ftr, int ftr_h, int ftr_w) {

    __shared__ float target_cache[CONV_TILE_SIZE][CONV_TILE_SIZE];
    __shared__ float filter_cache[CONV_MAX_FILTER_SIZE][CONV_MAX_FILTER_SIZE];
    __shared__ float image_cache[CONV_TILE_SIZE + CONV_MAX_FILTER_SIZE - 1][CONV_TILE_SIZE + CONV_MAX_FILTER_SIZE - 1];

    const int target_h = im_h - ftr_h + 1;
    const int target_w = im_w - ftr_w + 1;
    const int target_c_size = target_h * target_w;
    const int target_im_size = target_c_size * n_ftr;

    const int image_c_size = im_h * im_w;
    const int image_im_size = image_c_size * c;

    const int ftr_c_size = ftr_h * ftr_w;
    const int ftr_im_size = ftr_c_size * c;

    const int n_blocks_w = (target_w + blockDim.x - 1) / blockDim.x;
    const int n_blocks_h = (target_h + blockDim.y - 1) / blockDim.y;
    const int n_blocks_c = n_blocks_h * n_blocks_w;
    const int n_blocks_im = n_blocks_c * n_ftr;

    int bid = blockIdx.x;

    // loop over all output blocks
    while (bid < n * n_blocks_im) {
        int b_n = bid / n_blocks_im;
        int b_c = (bid % n_blocks_im) / n_blocks_c;
        int b_h = (bid % n_blocks_c) / n_blocks_w;
        int b_w = bid % n_blocks_w;

        int i_base_h = b_h * blockDim.y;
        int i_base_w = b_w * blockDim.x;

        // reset all targests
        // __syncthreads();
        target_cache[threadIdx.y][threadIdx.x] = 0;

        // loop over input channels
        for (int k = 0; k < c; k++) {
            int i_base = b_n * image_im_size + k * image_c_size;
            int f_base = b_c * ftr_im_size + k * ftr_c_size;

            // load data cache first

            for (int h = threadIdx.y; h < blockDim.y + ftr_h - 1 && h < im_h; h += blockDim.y)
                for (int w = threadIdx.x; w < blockDim.x + ftr_w - 1 && w < im_w; w += blockDim.x)
                    image_cache[h][w] = image[i_base + (i_base_h + h) * im_w + (i_base_w + w)];
            // __syncthreads();

            for (int h = threadIdx.y; h < ftr_h; h += blockDim.y)
                for (int w = threadIdx.x; w < ftr_w; w += blockDim.x)
                    filter_cache[h][w] = filter[f_base + h * ftr_w + w];
            __syncthreads();

            if (i_base_h + threadIdx.y < target_h && i_base_w + threadIdx.x < target_w) {
                float s = 0;
                for (int h = 0; h < ftr_h; h++)
                    for (int w = 0; w < ftr_w; w++)
                        s += image_cache[threadIdx.y + h][threadIdx.x + w] * filter_cache[h][w];
                target_cache[threadIdx.y][threadIdx.x] += s;
            }
            __syncthreads();
        }

        // write to output
        if (i_base_h + threadIdx.y < target_h && i_base_w + threadIdx.x < target_w)
            target[b_n * target_im_size + b_c * target_c_size + (i_base_h + threadIdx.y) * target_w + (i_base_w + threadIdx.x)] \
                = target_cache[threadIdx.y][threadIdx.x];

        bid += blockDim.x;
    }
}

__global__ void kConvolve(float* image, float* filter, float* target,
        int n, int c, int im_h, int im_w, int ftr_h, int ftr_w, int pad_h, int pad_w, int stride_h, int stride_w) {
    
}

